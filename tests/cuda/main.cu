#include <stdio.h>
#include <hip/hip_runtime.h>
__global__ void kernel() {
    printf("Hello from GPU!\n");
}

int main() {
    kernel<<<1, 1>>>();
    hipDeviceSynchronize();
    return 0;
}